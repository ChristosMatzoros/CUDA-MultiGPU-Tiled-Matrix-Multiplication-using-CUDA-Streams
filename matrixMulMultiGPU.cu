/*********************
MIT License

Copyright (c) 2020 Matzoros Christos

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

***********************/

/*********************
The purpose of this code is to execute multi-GPU matrix multiplication with multiple kernel 
invocations using streams. The program split the computation into 4 individual computations 
as it is shown below. The proportion of the size of the block is variable.
  
  ------------------------------------------------
    A * B = C   

    |  A1  |     |    |    |       C1 | C2
    -------- *   | B1 | B2 |   =   -------
    |  A2  |     |    |    |       C3 | C4 
  
    A1 * B1 = C1
    A1 * B2 = c2
    A2 * B1 = C3
    A2 * B2 = C4
    
    These 4 computations may take place simultaneously on 4 different GPUs.
  ------------------------------------------------

***********************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
// CUDA runtime
//#include <cuda_runtime.h>


//Error handling using functions of the CUDA runtime API
#define cudaCheckError() {                                                              \
    hipError_t e=hipGetLastError();                                                   \
    if(e!=hipSuccess) {                                                                \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));   \
        hipDeviceReset();                                                              \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

//This macro checks malloc() and cudaMalloc() return values
#define Check_Allocation_Return_Value(a){   \
    if(a==NULL) {                           \
    printf("Allocation Error\n");           \
    hipDeviceReset();                      \
    exit(EXIT_FAILURE);                     \
    }                                       \
}


//general kernel(not used)
__global__ void matrix_multiplication(double *A,double *B,double *C,int width){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((idy>=width)||((idx>=width))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*width+idx];
    }
    
    C[idy*width+idx] = prod_val;
}

// Kernel for the computation of C1 portion
__global__ void kernelC1(double *A,double *B,double *C,int width, double r){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int step;
    double prod_val = 0;
    
    if((idy>=(int)(width*r))||(idx>=(int)(width*r))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*(int)(width*r)+idx];
    }
    
    C[idy*(int)(width*r)+idx] = prod_val;
}

// Kernel for the computation of C2 portion
__global__ void kernelC2(double *A,double *B,double *C,int width, double r){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int step;
    double prod_val = 0;
    
    if((idy>=(int)(width*r))||(idx>=(int)(width*(1-r)))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*(int)(width*(1-r))+idx];
    }
    
    C[idy*(int)(width*(1-r))+idx] = prod_val;
}


// Kernel for the computation of C3 portion
__global__ void kernelC3(double *A,double *B,double *C,int width, double r){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((idy>=(int)(width*(1-r)))||(idx>=(int)(width*r))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*(int)(width*r)+idx];
    }
    
    
    C[idy*(int)(width*r)+idx] = prod_val;
}

// // Kernel for the computation of C4 portion
__global__ void kernelC4(double *A,double *B,double *C,int width, double r){
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((idy>=(int)(width*(1-r)))||(idx>=(int)(width*(1-r)))) return;
    
    for(step=0;step<width;step++){
        prod_val += A[idy*width+step] * B[step*(int)(width*(1-r))+idx];
    }
    C[idy*(int)(width*(1-r))+idx] = prod_val;
}



int main(int argc,char *argv[]){
    const int num_streams = 4;
    hipStream_t streams[num_streams];
    int N =7000;
    double *hA,*hB,*hC;
    int id,j,i;
    int ndev;
    double r = 0.5;
    double inv_r = (1-r);
    double *hA1,*hA2,*hB1,*hB2,*hC1,*hC2,*hC3,*hC4;
    double *dA1,*dA1_2,*dA2,*dA2_2,*dB1,*dB1_2,*dB2,*dB2_2;
    double *dC1,*dC2,*dC3,*dC4;
    
    printf("\nNumber of elements of the final matrix: %d\n",N * N);
    printf("Block 1 width: %d\n",(int)(N*r));
    printf("Block 2 width: %d\n",(int)(N*inv_r));
         
    hipGetDeviceCount(&ndev);
    if(ndev==0){
        printf("NO GPU DEVICES AVAILABLE\n\n");
        exit(-1);
            
    }else{
        printf("Number of available GPUs: %d\n\n",ndev);
    }
        
    hipHostMalloc(&hA,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA)
    hipHostMalloc(&hB,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB)
    hipHostMalloc(&hC,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC)
    memset(hC,0,N*N*sizeof(double));
        
    srand (time(NULL));
    
    for(i=0;i<N*N;i++){
        hA[i] = rand()%10;
        hB[i] = rand()%10;
    }
    
    //Grid and block size initialization
    int grid_width = 1+N/32;
    dim3 dimGrid(grid_width,grid_width,1);
    dim3 dimBlock(32,32,1);
    
        
    ////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////
    // kernel 1
    id=0;
    hipSetDevice((int)(id%ndev));
    //cudaStreamCreate(&streams[id]);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    
    hipHostMalloc(&hA1,(int)(N*N*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA1)
    hipHostMalloc(&hB1,(int)(N*N*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB1)
    hipHostMalloc(&hC1,(int)(N*N*r*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC1)
    
    for(int i=0;i<(int)(N*r);i++){
        for(int j=0;j<N;j++){
            hA1[i*N+j] =  hA[i*N+j];
        }
    }
    for(int i=0;i<N;i++){
        for(int j=0;j<(N*r);j++){
            hB1[i*(int)(N*r)+j] =  hB[i*N+j];
        }
    }

    hipMalloc((void**)&dA1,(int)(N*N*r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dB1,(int)(N*N*r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dC1,(int)(N*N*r*r*sizeof(double)));
    cudaCheckError()
        
    // kernel 2
    id=1;
    hipSetDevice((int)(id%ndev));
    //cudaStreamCreate(&streams[id]);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    
    hipHostMalloc(&hB2,(int)(N*N*inv_r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB2)
    hipHostMalloc(&hC2,(int)(N*N*r*inv_r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC2)
    
    for(int i=0;i<N;i++){
        for(int j=0;j<(N*inv_r);j++){
            hB2[i*(int)(N*inv_r)+j] =  hB[i*N+(int)(N*r)+j];
        }
    }
     
    hipMalloc((void**)&dA1_2,(int)(N*N*r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dB2,(int)(N*N*inv_r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dC2,(int)(N*N*r*inv_r*sizeof(double)));
    cudaCheckError()
        
    // kernel 3
    id=2;
    hipSetDevice(id%ndev);
    //cudaStreamCreate(&streams[id]);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    
    hipHostMalloc(&hA2,(int)(N*N*inv_r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA2)
    hipHostMalloc(&hC3,(int)(N*N*inv_r*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC3)
    
    for(int i=0;i<(int)(N*inv_r);i++){
        for(int j=0;j<N;j++){
            hA2[i*N+j] =  hA[(i+(int)(N*r))*N+j];
        }
    }
    
    hipMalloc((void**)&dA2,(int)(N*N*inv_r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dB1_2,(int)(N*N*r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dC3,(int)(N*N*r*inv_r*sizeof(double)));
    cudaCheckError()  
        
    // kernel 4
    id=3;
    hipSetDevice(id%ndev);
    //cudaStreamCreate(&streams[id]);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);

    hipHostMalloc(&hC4,(int)(N*N*inv_r*inv_r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC4)
    
    hipMalloc((void**)&dA2_2,(int)(N*N*inv_r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dB2_2,(int)(N*N*inv_r*sizeof(double)));
    cudaCheckError()
    hipMalloc((void**)&dC4,(int)(N*N*inv_r*inv_r*sizeof(double)));
    cudaCheckError()
        
    //////////////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////////////
   
    printf("CPU-->GPU Memory copy(A1,B1,C1) - cudaMemcpyAsync\n");
    
    id=0;
    hipSetDevice(id%ndev);
        
    hipMemcpyAsync(dA1,hA1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    hipMemcpyAsync(dB1,hB1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
        
    printf("Kernel 1 Execution...\n");
    kernelC1 <<< dimGrid,dimBlock,0,streams[id]>>>(dA1,dB1,dC1,N,r);
    cudaCheckError()
    
    ///////////////////////////////////////////////////////////////////////////////  
    
    id=1;
    hipSetDevice(id%ndev);
    
    printf("CPU-->GPU Memory copy(A1,B2,C2) - cudaMemcpyAsync\n");
    hipMemcpyAsync(dA1_2,hA1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    hipMemcpyAsync(dB2,hB2,(int)(N*N*inv_r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    
    printf("Kernel 2 Execution...\n");
    kernelC2 <<< dimGrid,dimBlock,0,streams[id]>>>(dA1_2,dB2,dC2,N,r);
    cudaCheckError()
    
    ///////////////////////////////////////////////////////////////////////////////
    
    id=2;
    hipSetDevice(id%ndev);
    
    printf("CPU-->GPU Memory copy(A2,B1,C3) - cudaMemcpyAsync\n");
    hipMemcpyAsync(dA2,hA2,(int)(N*N*inv_r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    hipMemcpyAsync(dB1_2,hB1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    
    printf("Kernel 3 Execution...\n");
    kernelC3 <<< dimGrid,dimBlock,0,streams[id]>>>(dA2,dB1_2,dC3,N,r);
    cudaCheckError()

    ///////////////////////////////////////////////////////////////////////////////
    
    id=3;
    hipSetDevice(id%ndev);
    
    printf("CPU-->GPU Memory copy(A2,B2,C4) - cudaMemcpyAsync\n");
    hipMemcpyAsync(dA2_2,hA2,(int)(N*N*inv_r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()
    hipMemcpyAsync(dB2_2,hB2,(int)(N*N*inv_r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    cudaCheckError()

    printf("Kernel 4 Execution...\n");
    kernelC4 <<< dimGrid,dimBlock,0,streams[id]>>>(dA2_2,dB2_2,dC4,N,r);
    cudaCheckError()

    
    ///////////////////////////////////////////////////////////////////////////////
    printf("GPU-->CPU Memory copy (dC1) - cudaMemcpyAsync\n");
    hipMemcpyAsync(hC1,dC1,(int)(N*N*r*r*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    cudaCheckError()
    
    printf("GPU-->CPU Memory copy (dC2) - cudaMemcpyAsync\n");
    hipMemcpyAsync(hC2,dC2,(int)(N*N*r*inv_r*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    cudaCheckError()
    
    printf("GPU-->CPU Memory copy (dC3) - cudaMemcpyAsync\n");
    hipMemcpyAsync(hC3,dC3,(int)(N*N*r*inv_r*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    cudaCheckError()
    
    printf("GPU-->CPU Memory copy (dC4) - cudaMemcpyAsync\n");
    hipMemcpyAsync(hC4,dC4,(int)(N*N*inv_r*inv_r*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    cudaCheckError()
    
    
    //Synchronize in order to process the results of every invocation

    id=0;
    hipSetDevice(id%ndev);
    hipStreamSynchronize(streams[id]);
    
    id=1;
    hipSetDevice(id%ndev);
    hipStreamSynchronize(streams[id]);

    id=2;
    hipSetDevice(id%ndev);
    hipStreamSynchronize(streams[id]);

    id=3;
    hipSetDevice(id%ndev);
    hipStreamSynchronize(streams[id]);

    //create the final Matrix
    for(i=0;i<(int)N*r;i++){
        for(j=0;j<(int)N*r;j++){
              hC[i*N+j] = hC1[i*(int)(N*r)+j];
              //printf("hC[%d]:%f ",i*N+j,hC[i*N+j]);
        }
        //printf("\n");
    }
    //printf("\n");
    
    
    for(i=0;i<(int)N*r;i++){
        for(j=0;j<(int)(N*inv_r);j++){
             hC[i*N+j+(int)(N*r)] = hC2[i*(int)(N*inv_r)+j];
             //printf("hC[%d]:%f",i*N+j+(int)(N*r),hC[i*N+j+(int)(N*r)]);
        }
        //printf("\n");
    }
    //printf("\n");
    
    for(i=0;i<(int)(N*inv_r);i++){
        for(j=0;j<(int)(N*r);j++){
             hC[(i+(int)(N*r))*N+j] = hC3[i*(int)(N*r)+j];
             //printf("hC[%d]:%f",(i+(int)(N*r))*N+j,hC[(i+(int)(N*r))*N+j]);
        }
        //printf("\n");
    }
    //printf("\n"); 
    
  
    for(i=0;i<(int)(N*inv_r);i++){
        for(j=0;j<(int)(N*inv_r);j++){
            hC[(i+(int)(N*r))*N+j+(int)(N*r)] = hC4[i*(int)(N*inv_r)+j];
          //  printf("hC[%d]:%f",(i+(int)(N*r))*N+j+(int)(N*r),hC[(i+(int)(N*r))*N+j+(int)(N*r)]);
        }
       // printf("\n");
    }
  //  printf("\n"); 
    
    
    /*
    //Compare the GPU result with CPU computation(for validation)
    printf("Check results...\n");
    int k;
    double res; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            res=0;
            for(k=0;k<N;k++){
                res+=hA[i*N+k]*hB[k*N+j];
            }
            
           //printf("%8.3f ",res);
           if(res != hC[i*N+j]){
                printf("NOT OK i:%d, j:%d\n",i,j);
                printf("true value:%f - computed value:%f\n\n",res,hC[i*N+j]);
           }
        }
        //printf("\n");
    }
    */
    
    
    
    printf("Free Host and Device Memory\n");
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipHostFree(hA1);
    hipHostFree(hA2);
    hipHostFree(hB1);
    hipHostFree(hB2);
    hipHostFree(hC1);
    hipHostFree(hC2);
    hipHostFree(hC3);
    hipHostFree(hC4);
    
    id=0;
    hipSetDevice(id%ndev);
    hipFree(dA1);
    cudaCheckError()
    hipFree(dB1);
    cudaCheckError()
    hipFree(dC1);
    cudaCheckError()
    
    id=1;
    hipSetDevice(id%ndev);
    hipFree(dA1_2);
    cudaCheckError()
    hipFree(dB2);
    cudaCheckError()
    hipFree(dC2);
    cudaCheckError()
    
    id=2;
    hipSetDevice(id%ndev);
    hipFree(dA2);
    cudaCheckError()
    hipFree(dB1_2);
    cudaCheckError()
    hipFree(dC3);
    cudaCheckError()
    
    id=3;
    hipSetDevice(id%ndev);
    hipFree(dA2_2);
    cudaCheckError()
    hipFree(dB2_2);
    cudaCheckError()
    hipFree(dC4);
    cudaCheckError()
    
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
    hipStreamDestroy(streams[2]);
    hipStreamDestroy(streams[3]);
    
    return(0);
}
